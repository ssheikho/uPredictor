
#include <hip/hip_runtime.h>
__global__ void gpu_KIDepthToVertices(const float *depthIn,
                                    float4 * vertOut, int *segMap,
                                    const int width,
                                    const int height,
                                    const float2 pp,
                                    const float2 fl) {

    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;
    const int index = u + v*width;

    if (u >= width || v >= height)
        return;

    float depth = depthIn[index];// / 1000.0;
    vertOut[index] = make_float4( (u - pp.x)*(depth/fl.x),
                                  (v - pp.y)*(depth/fl.y),
                                  depth,
                                 //segMap[index] == 20 ? 1.0f : 0.0f);
                                 1.0f);

}

void KIDepthToVertices(const float *depthIn, float4 *vertOut, int *segMap
	, const int width, const int height, const float2 pp, const float2 fl) {
    dim3 block(16,8,1);
    dim3 grid( ceil( width / (float)block.x), ceil( height / (float)block.y ));

    gpu_KIDepthToVertices<<<grid,block>>>(depthIn, vertOut, segMap
		, width, height, pp, fl);
}

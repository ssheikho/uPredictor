#include "hip/hip_runtime.h"
#define _EXPORTING
#include "UBCColorTrackerCUDA.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstring>
#include <vector>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void cloudNormalKernel(
	CameraSpacePoint *camPts
	, float *outVectsX, float *outVectsY
	, float *outNormsX, float *outNormsY
	, int dimX, int dimY) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int maxIdx = dimX * dimY;
	int nextX = idx + 1;
	int nextY = idx + dimX;

	int idx3 = idx * 3;

	if (nextX % dimX != 0) {
		float x = outVectsX[idx3] = (camPts[nextX].X - camPts[idx].X);
		float y = outVectsX[idx3 + 1] = (camPts[nextX].Y - camPts[idx].Y);
		float z = outVectsX[idx3 + 2] = (camPts[nextX].Z - camPts[idx].Z);

		outNormsX[idx] = sqrt(x * x + y * y + z * z);

		outVectsX[idx3] /= outNormsX[idx];
		outVectsX[idx3 + 1] /= outNormsX[idx];
		outVectsX[idx3 + 2] /= outNormsX[idx];
	}

	if (nextY < maxIdx) {
		float x = outVectsY[idx3] = (camPts[nextY].X - camPts[idx].X);
		float y = outVectsY[idx3 + 1] = (camPts[nextY].Y - camPts[idx].Y);
		float z = outVectsY[idx3 + 2] = (camPts[nextY].Z - camPts[idx].Z);

		outNormsY[idx] = sqrt(x * x + y * y + z * z);

		outVectsY[idx3] /= outNormsY[idx];
		outVectsY[idx3 + 1] /= outNormsY[idx];
		outVectsY[idx3 + 2] /= outNormsY[idx];
	}
}

void setupCUDA() {}

int setDevice() {
	hipError_t cudaStatus = hipSetDevice(0);
	return 0;
}

__global__ void marginalSegmentationKernel(
	unsigned char *inputImage
	, unsigned char *outputImage
	, int *outputSegments
	, MarginalTrackerParams *params) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idxThreeX = 3 * idx;
	if (
		((inputImage[idxThreeX + params->_mostlyChan] -
		inputImage[idxThreeX + params->_fromChanA]) > params->_marginA) &&
		((inputImage[idxThreeX + params->_mostlyChan] -
		inputImage[idxThreeX + params->_fromChanB]) > params->_marginB)
		) {
		outputImage[idxThreeX] = params->_indicColorB;
		outputImage[idxThreeX + 1] = params->_indicColorG;
		outputImage[idxThreeX + 2] = params->_indicColorR;
		outputSegments[idx] = 20;
	}
	else {
		outputImage[idxThreeX] = inputImage[idxThreeX];
		outputImage[idxThreeX + 1] = inputImage[idxThreeX + 1];
		outputImage[idxThreeX + 2] = inputImage[idxThreeX + 2];
		outputSegments[idx] = -1;
	}
}

CUDATracker setupCUDATracker(
	struct MarginalTrackerParams *marginalTrackerParams, int stream) {
	hipError_t cudaStatus;

	CUDATracker tracker;

	cudaStatus = hipMalloc((void**)&tracker._devInputImage
		, marginalTrackerParams->_sizeII);
	cudaStatus = hipMalloc((void**)&tracker._devOutputImage
		, marginalTrackerParams->_sizeII);
	cudaStatus = hipMalloc((void**)&tracker._devOutputSegments
		, marginalTrackerParams->_npix * sizeof(int));
	cudaStatus = hipMalloc((void**)&tracker._devMarginalTrackerParams
		, sizeof(MarginalTrackerParams));

	cudaStatus = hipMemcpy(tracker._devMarginalTrackerParams, marginalTrackerParams
		, sizeof(MarginalTrackerParams), hipMemcpyHostToDevice);

	tracker._marginalTrackerParams = marginalTrackerParams;

	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize,
		&tracker._blockSize,
		marginalSegmentationKernel, 0, 0);

	tracker._gridSize =
		(marginalTrackerParams->_npix + tracker._blockSize - 1)
		/ tracker._blockSize;
	
	return tracker;
}

int doMarginalSegmentation(
	struct CUDATracker *ct
	, unsigned char *inputImage
	, unsigned char *outputImage
	, int *outputSegments) {
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(ct->_devInputImage, inputImage
		, ct->_marginalTrackerParams->_sizeII, hipMemcpyHostToDevice);

	cudaStatus =
		hipMemcpy(ct->_devMarginalTrackerParams, ct->_marginalTrackerParams
		, sizeof(MarginalTrackerParams), hipMemcpyHostToDevice);

	marginalSegmentationKernel << <ct->_gridSize, ct->_blockSize >> >
		(ct->_devInputImage, ct->_devOutputImage, ct->_devOutputSegments
		, ct->_devMarginalTrackerParams);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = hipMemcpy(outputImage, ct->_devOutputImage
		, ct->_marginalTrackerParams->_sizeII, hipMemcpyDeviceToHost);

	cudaStatus = hipMemcpy(outputSegments, ct->_devOutputSegments
		, ct->_marginalTrackerParams->_npix * sizeof(int)
		, hipMemcpyDeviceToHost);

	return 0;
}

__global__ void cloudSineKernel(
	CameraSpacePoint *camPts
	, float *inVectsX, float *inVectsY
	, float *outSinesX, float *outSinesY
	, float *outCosinesX, float *outCosinesY
	, int dimX, int dimY) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int maxIdx = dimX * dimY;
	int nextX = idx + 1;
	int nextY = idx + dimX;

	int idx3 = idx * 3;

	if (nextX < maxIdx) {
		if ((nextX % dimX) > 0) {
			int nextX3 = nextX * 3;

			float aX = inVectsX[idx3];
			float aY = inVectsX[idx3 + 1];
			float aZ = inVectsX[idx3 + 2];

			float bX = inVectsX[nextX3];
			float bY = inVectsX[nextX3 + 1];
			float bZ = inVectsX[nextX3 + 2];

			float cX = aZ * bY - aY * bZ;
			float cY = aX * bZ - aZ * bX;
			float cZ = aY * bX - aX * bY;

			outSinesX[idx] = aX;// *bX + aY * bY + aZ * bZ;
			outCosinesX[idx] = sqrt(cX + cY + cZ);

		}
		else {
			outSinesX[idx] = 0;
			outCosinesX[idx] = 0;
		}

		if (nextY < maxIdx) {
			int nextY3 = nextY * 3;

			float aX = inVectsY[idx3];
			float aY = inVectsY[idx3 + 1];
			float aZ = inVectsY[idx3 + 2];

			float bX = inVectsY[nextY3];
			float bY = inVectsY[nextY3 + 1];
			float bZ = inVectsY[nextY3 + 2];

			float cX = aZ * bY - aY * bZ;
			float cY = aX * bZ - aZ * bX;
			float cZ = aY * bX - aX * bY;

			outSinesY[idx] = aX;// *bX + aY * bY + aZ * bZ;
			outCosinesY[idx] = sqrt(cX + cY + cZ);
		}
		else {
			outSinesY[idx] = 0;
			outCosinesY[idx] = 0;
		}
	}
	else {
		outSinesX[idx] = 0;
		outSinesY[idx] = 0;
		outCosinesX[idx] = 0;
		outCosinesY[idx] = 0;
	}
}

CUDAComputeCloudNormals setupCloudNormalComp(int dimX, int dimY, int stream) {
	hipError_t cudaStatus;
	CUDAComputeCloudNormals ccn;
	ccn._dimX = dimX;
	ccn._dimY = dimY;
	ccn._pts = dimX * dimY;

	cudaStatus = hipMalloc((void**)&ccn._devCamPts, ccn._pts * sizeof(CameraSpacePoint));

	cudaStatus = hipMalloc((void**)&ccn._devOutVectsX, ccn._pts * 3 * sizeof(float));
	cudaStatus = hipMalloc((void**)&ccn._devOutVectsY, ccn._pts * 3 * sizeof(float));

	cudaStatus = hipMalloc((void**)&ccn._devOutNormsX, ccn._pts * sizeof(float));
	cudaStatus = hipMalloc((void**)&ccn._devOutNormsY, ccn._pts * sizeof(float));

	cudaStatus = hipMalloc((void**)&ccn._devSineX, ccn._pts * sizeof(float));
	cudaStatus = hipMalloc((void**)&ccn._devSineY, ccn._pts * sizeof(float));

	cudaStatus = hipMalloc((void**)&ccn._devCosineX, ccn._pts * sizeof(float));
	cudaStatus = hipMalloc((void**)&ccn._devCosineY, ccn._pts * sizeof(float));

	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize,
		&ccn._blockSizeN,
		cloudNormalKernel, 0, 0);

	ccn._gridSizeN = (ccn._pts + ccn._blockSizeN - 1) / ccn._blockSizeN;

	hipOccupancyMaxPotentialBlockSize(&minGridSize,
		&ccn._blockSizeS,
		cloudSineKernel, 0, 0);

	ccn._gridSizeS = (ccn._pts + ccn._blockSizeS - 1) / ccn._blockSizeS;

	//hipCtxPopCurrent(&_contexts[ccn._stream]);

	//ReleaseMutex(_cudaMutex);

	return ccn;
}

void computeCloudNormals(
	struct CUDAComputeCloudNormals *ct
	, CameraSpacePoint *camPts
	, float *outVectsX, float *outVectsY
	, float *outNormsX, float *outNormsY
	, float *outSinesX, float *outSinesY
	, float *outCosinesX, float *outCosinesY) {
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(ct->_devCamPts, camPts
		, ct->_pts * sizeof(CameraSpacePoint), hipMemcpyHostToDevice);
	
	//, 0, _streams[ct->_stream]
	cloudNormalKernel << <ct->_gridSizeN, ct->_blockSizeN >> > (
		ct->_devCamPts
		, ct->_devOutVectsX, ct->_devOutVectsY
		, ct->_devOutNormsX, ct->_devOutNormsY
		, ct->_dimX, ct->_dimY);

	cudaStatus = hipDeviceSynchronize();

	//, 0, _streams[ct->_stream]
	cloudSineKernel << <ct->_gridSizeS, ct->_blockSizeS>> > (
		ct->_devCamPts
		, ct->_devOutVectsX, ct->_devOutVectsY
		, ct->_devSineX, ct->_devSineY
		, ct->_devCosineX, ct->_devCosineY
		, ct->_dimX, ct->_dimY);

	cudaStatus = hipDeviceSynchronize();
	
	cudaStatus = hipMemcpy(outVectsX, ct->_devOutVectsX
		, ct->_pts * 3 * sizeof(float), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(outVectsY, ct->_devOutVectsY
		, ct->_pts * 3 * sizeof(float), hipMemcpyDeviceToHost);

	cudaStatus = hipMemcpy(outNormsX, ct->_devOutNormsX
		, ct->_pts * sizeof(float), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(outNormsY, ct->_devOutNormsY
		, ct->_pts * sizeof(float), hipMemcpyDeviceToHost);

	cudaStatus = hipMemcpy(outSinesX, ct->_devSineX
		, ct->_pts * sizeof(float), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(outSinesY, ct->_devSineY
		, ct->_pts * sizeof(float), hipMemcpyDeviceToHost);

	cudaStatus = hipMemcpy(outCosinesX, ct->_devCosineX
		, ct->_pts * sizeof(float), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(outCosinesY, ct->_devCosineY
		, ct->_pts * sizeof(float), hipMemcpyDeviceToHost);

	//hipCtxPopCurrent(&_contexts[ct->_stream]);

	//ReleaseMutex(_cudaMutex);
}

__global__ void floatImageKernel(float *devFloat, char *devImage, int nChans
	, int nPix) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idx3 = idx * 3;
	char v = ceil(255.0 * devFloat[idx]);
	devImage[idx3] = v;
	devImage[idx3 + 1] = v;
	devImage[idx3 + 2] = v;
}

CUDAComputeFloatToImage setupCUDAComputeFloatImage(int nPix, int nChans) {
	//WaitForSingleObject(_cudaMutex, INFINITE);

	hipError_t cudaStatus;

	CUDAComputeFloatToImage trac;
	trac._nChans = nChans;
	trac._nPix = nPix;

	//Allocate buffers on GPU for parameters
	cudaStatus = hipMalloc((void**)&trac._devFloat, nPix * sizeof(float));
	cudaStatus = hipMalloc((void**)&trac._devImage, nPix * 3 * sizeof(char));

	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize,
		&trac._blockSize,
		floatImageKernel, 0, 0);

	trac._gridSize = (nPix + trac._blockSize - 1) / trac._blockSize;

	//hipFree(trac._devFloat);
	//hipFree(trac._devImage);

	//ReleaseMutex(_cudaMutex);

	return trac;
}

void computeFloatImage(float *inFloat, char *outImg, CUDAComputeFloatToImage *cccn) {
	//WaitForSingleObject(_cudaMutex, INFINITE);

	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(cccn->_devFloat, inFloat
		, cccn->_nPix * sizeof(float), hipMemcpyHostToDevice);

	floatImageKernel << <cccn->_gridSize, cccn->_blockSize >> >(
		cccn->_devFloat, cccn->_devImage, cccn->_nChans, cccn->_nPix);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = hipMemcpy(outImg, cccn->_devImage
		, cccn->_nPix * 3 * sizeof(char), hipMemcpyDeviceToHost);

	//for (int i = 0; i < 10000; i++) outImg[i] = 200;

	//ReleaseMutex(_cudaMutex);
}
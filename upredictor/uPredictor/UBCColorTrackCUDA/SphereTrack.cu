#include "hip/hip_runtime.h"
#define _EXPORTING
#include "UBCColorTrackerCUDA.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstring>
#include <vector>

__global__ void resizeGridKernel(
	CameraSpacePoint *centers
	, struct GridParams *gp
	) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < gp->_nGridCells) {
		int xPos = idx % gp->_gridDim;
		int idxModX = (idx - xPos) / gp->_gridDim;
		int yPos = idxModX % gp->_gridDim;
		int zPos = (idxModX - yPos) / gp->_gridDim;

		centers[idx].X = gp->_xFirstCenter +((double)xPos * gp->_xSliceWidth);
		centers[idx].Y = gp->_yFirstCenter +((double)yPos * gp->_ySliceWidth);
		centers[idx].Z = gp->_zFirstCenter +((double)zPos * gp->_zSliceWidth);
	}

}

__global__ void computeCostsKernel (
	CameraSpacePoint *centers
	, CameraSpacePoint *pts
	, int *inputSegments
	, int segment
	, float *costs
	, int *assignments
	, struct GridParams *gp
	) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < gp->_nPts) {
		if (inputSegments[idx] == segment) {
			int assignment = 0;
			float xx = centers[0].X - pts[idx].X;
			float yy = centers[0].Y - pts[idx].Y;
			float zz = centers[0].Z - pts[idx].Z;
			float cost = fabs(sqrt(xx + yy + zz) - gp->_r);
			for (int i = 1; i < gp->_nGridCells; i++) {
				xx = centers[i].X - pts[idx].X;
				yy = centers[i].Y - pts[idx].Y;
				zz = centers[i].Z - pts[idx].Z;
				float curcost = fabs(sqrt(xx + yy + zz) - gp->_r);
				if (curcost < cost) {
					cost = curcost;
					assignment = i;
				}
			}
			costs[idx] = cost;
			assignments[idx] = assignment;
		}
		else {
			costs[idx] = -1.0f;
			assignments[idx] = -1;
		}
	}
}

CUDASphereDetector setupCUDAGridFilter(GridParams *gp) {
	hipError_t cudaStatus;

	CUDASphereDetector retVal;
	retVal._hostGridParams = gp;
	gp->_nGridCells = gp->_gridDim * gp->_gridDim * gp->_gridDim;

	retVal._hostCenters = new CameraSpacePoint[gp->_nGridCells];
	retVal._hostCosts = new float[gp->_nGridCells];
	retVal._hostSphereAssignments = new int[gp->_nPts];

	cudaStatus = hipMalloc((void**)&retVal._devPts
		, gp->_nPts * sizeof(CameraSpacePoint));
	cudaStatus = hipMalloc((void**)&retVal._devCenters
		, gp->_nGridCells * sizeof(CameraSpacePoint));
	cudaStatus = hipMalloc((void**)&retVal._devCosts
		, gp->_nGridCells * sizeof(float));
	cudaStatus = hipMalloc((void**)&retVal._devSphereAssignments
		, gp->_nPts * sizeof(int));
	cudaStatus = hipMalloc((void**)&retVal._devInputSegments
		, gp->_nPts * sizeof(int));
	cudaStatus = hipMalloc((void**)&retVal._devGridParams
		, sizeof(GridParams));

	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &retVal._blockR
		, resizeGridKernel, 0, 0);

	retVal._gridR = (gp->_nGridCells + retVal._blockR - 1) / retVal._blockR;
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &retVal._blockCC
		, computeCostsKernel, 0, 0);

	retVal._gridCC = (gp->_nPts + retVal._blockCC - 1) / retVal._blockCC;

	return retVal;
}

//Compute diameter of a sphere
//double d = 2.0 * csd->_r;
void resizeGrid(
	float minX3D, float maxX3D,
	float minY3D, float maxY3D,
	float minZ3D, float maxZ3D,
	CUDASphereDetector *csd) {
	hipError_t cudaStatus;
	GridParams *hgp = csd->_hostGridParams;

	hgp->_d = 2.0 * hgp->_r;

	float xMaxSpan = (maxX3D + hgp->_d);
	float yMaxSpan = (maxY3D + hgp->_d);
	float zMaxSpan = (maxZ3D + hgp->_d);

	float xMinSpan = (minX3D - hgp->_d);
	float yMinSpan = (minY3D - hgp->_d);
	float zMinSpan = (minZ3D - hgp->_d);

	float xSpan = xMaxSpan - xMinSpan;
	float ySpan = yMaxSpan - yMinSpan;
	float zSpan = zMaxSpan - zMinSpan;

	hgp->_xSliceWidth = xSpan / (double)(hgp->_gridDim);
	hgp->_ySliceWidth = ySpan / (double)(hgp->_gridDim);
	hgp->_zSliceWidth = zSpan / (double)(hgp->_gridDim);

	hgp->_xFirstCenter = xMinSpan + hgp->_r;
	hgp->_yFirstCenter = yMinSpan + hgp->_r;
	hgp->_zFirstCenter = zMinSpan + hgp->_r;

	cudaStatus = hipMemcpy(csd->_devGridParams, csd->_hostGridParams
		, sizeof(GridParams), hipMemcpyHostToDevice);

	resizeGridKernel << <csd->_gridR, csd->_blockR >> >
		(csd->_devCenters, csd->_devGridParams);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = hipMemcpy(csd->_hostCenters, csd->_devCenters
		, hgp->_nGridCells
		* sizeof(CameraSpacePoint), hipMemcpyDeviceToHost);
}

void computeCosts(
	CameraSpacePoint *hostPts
	, int *hostInputSegments
	, int segment
	, CUDASphereDetector *csd
	) {
	GridParams *hgp = csd->_hostGridParams;

	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(csd->_devPts, hostPts
		, hgp->_nPts * sizeof(CameraSpacePoint), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(csd->_devInputSegments, hostInputSegments
		, hgp->_nPts * sizeof(int), hipMemcpyHostToDevice);

	computeCostsKernel << <csd->_gridCC, csd->_blockCC >> >
		(csd->_devCenters, csd->_devPts, csd->_devInputSegments, segment
			, csd->_devCosts, csd->_devSphereAssignments, csd->_devGridParams);
	cudaStatus = hipDeviceSynchronize();

	cudaStatus = hipMemcpy(csd->_hostCosts, csd->_devCosts
		, hgp->_nPts * sizeof(float), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(csd->_hostSphereAssignments, csd->_devSphereAssignments
		, hgp->_nPts * sizeof(int), hipMemcpyDeviceToHost);

}